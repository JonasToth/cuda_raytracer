#include "graphic/kernels/utility.h"
#include "graphic/kernels/shaded.h"
#include "management/window.h"
#include "management/world.h"

#include <thread>
#include <chrono>

static void raytrace_many_shaded(hipSurfaceObject_t surface, camera c,
                                 const triangle* triangles, std::size_t n_triangles,
                                 const light_source* lights, std::size_t n_lights)
{
    dim3 dimBlock(32,32);
    dim3 dimGrid((c.width() + dimBlock.x) / dimBlock.x,
                 (c.height() + dimBlock.y) / dimBlock.y);
    black_kernel<<<dimGrid, dimBlock>>>(surface, c.width(), c.height());
    std::clog << "Triangle ptr: " << triangles << "; " << n_triangles << std::endl
              << "LightSrc ptr: " << lights << "; " << n_lights << std::endl
              << "Surface     : " << surface << std::endl;
    trace_many_triangles_shaded<<<dimGrid, dimBlock>>>(surface, c,
                                                       triangles, n_triangles, 
                                                       lights, n_lights,
                                                       c.width(), c.height());
}

int main(int argc, char** argv)
{
    if(argc != 2)
    {
        std::cerr << "Warning: Give the ouputfile as argument, e.g. materials_smooth.png" 
                  << std::endl;
    }

    window win(800, 600, "Material Scene");
    auto w = win.getWindow();
    glfwMakeContextCurrent(w);

    // Camera Setup similar to blender
    camera c(win.getWidth(), win.getHeight(), 
             {0.0f, 0.5f, -2.0f}, {0.1f, 0.f, 1.f});
    surface_raii render_surface(win.getWidth(), win.getHeight());

    std::clog << "Setup Rendering Platform initialized" << std::endl;
    
    world_geometry scene("material_scene_smooth.obj");

    // Light Setup similar to blender (position and stuff taken from there)
    float spec[3] = {0.8f, 0.8f, 0.8f};
    float diff[3] = {0.8f, 0.8f, 0.8f};
    thrust::device_vector<light_source> lights(4);
    lights[0] = light_source{phong_light(spec, diff), coord{-1.4f, -1.4f, -1.4f}};
    lights[1] = light_source{phong_light(spec, diff), coord{ 1.4f, -1.4f, -1.4f}};
    lights[2] = light_source{phong_light(spec, diff), coord{-1.4f,  1.4f,  1.4f}};
    lights[3] = light_source{phong_light(spec, diff), coord{-1.4f, -1.4f,  1.4f}};

    std::clog << "World initialized" << std::endl;

    const auto& triangles = scene.triangles();
    raytrace_many_shaded(render_surface.getSurface(), c,
                         triangles.data().get(), triangles.size(),
                         lights.data().get(), lights.size());
    
    // seems necessary, otherwise the png is empty :/
    std::this_thread::sleep_for(std::chrono::milliseconds(2000));
    render_surface.render_gl_texture();

    if(argc == 2)
        render_surface.save_as_png(argv[1]);

    std::clog << "World rendered" << std::endl;

    return 0;
} 
