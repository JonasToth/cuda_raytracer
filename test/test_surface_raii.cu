#include "gtest/gtest.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include "surface_raii.h"


//TEST(visualization, basic_starting)
//{
    //vis(640, 480);
//}


int main(int argc, char** argv)
{
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
