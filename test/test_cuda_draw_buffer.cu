#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "camera.h"
#include "input_manager.h"
#include "macros.h"
#include "obj_io.h"
#include "triangle.h"
#include "ray.h"
#include "surface_raii.h"
#include "window.h"

#include <GLFW/glfw3.h>
#include <gsl/gsl>
#include <iostream>
#include <limits>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_new.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <utility>

const int Width = 800, Height = 800;
camera c(Width, Height, {2.f, 2.f, 2.f}, {0.f, 0.f, 1.f});

static void quit_with_q(GLFWwindow* w, int key, int scancode, int action, int mods)
{
    const float dP     = 0.5;
    const float dAngle = M_PI / 180. * 5.;

    auto& im = input_manager::instance();

    if(action == GLFW_PRESS)
        im.press(key);
    else if(action == GLFW_RELEASE)
        im.release(key);

    if(im.isPressed(GLFW_KEY_ESCAPE))
        glfwSetWindowShouldClose(w, GLFW_TRUE);
    else if(im.isPressed(GLFW_KEY_A))
        c.move({-dP, 0.f, 0.f});
    else if(im.isPressed(GLFW_KEY_D))
        c.move({dP, 0.f, 0.f});
    else if(im.isPressed(GLFW_KEY_W))
        c.move({0.f, 0.f, dP});
    else if(im.isPressed(GLFW_KEY_S))
        c.move({0.f, 0.f, -dP});
    else if(im.isPressed(GLFW_KEY_Q))
        c.move({0.f, dP, 0.f});
    else if(im.isPressed(GLFW_KEY_E))
        c.move({0.f, -dP, 0.f});
    else if(im.isPressed(GLFW_KEY_LEFT))
        c.swipe(0.f, -dAngle, 0.f);
    else if(im.isPressed(GLFW_KEY_RIGHT))
        c.swipe(0.f, dAngle, 0.f);
    else if(im.isPressed(GLFW_KEY_UP))
        c.swipe(dAngle, 0.f, 0.f);
    else if(im.isPressed(GLFW_KEY_DOWN))
        c.swipe(-dAngle, 0.f, 0.f);
    else
        return;

    std::clog << "Camera Position: " << c.origin() << std::endl;
    std::clog << "Camera Steering At: " << c.steering() << std::endl << std::endl;
}

static void control_steering(GLFWwindow* w, double xpos, double ypos)
{
    // xpos = alpha
    // ypos = beta
    //std::clog << "X: " << xpos << ";Y: " << ypos << std::endl;
    //float beta  = 2. * M_PI * xpos / Width;
    //float gamma = M_PI * ypos / Height;
    //c.swipe(beta, gamma);
    //std::clog << "Camera Steering At: " << c.steering() << std::endl;
}


__global__ void grayKernel(hipSurfaceObject_t Surface, int width, int height, float t)
{
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < width && y < height)
    {
        uchar4 Color;
        char new_t = t;
        Color.x = x - new_t;
        Color.y = y + new_t;
        Color.z = new_t;
        Color.w = 255;
        surf2Dwrite(Color, Surface, x * 4, y);
    }
}

void invokeRenderingKernel(hipSurfaceObject_t& Surface, float t)
{
    //std::clog << "Rendering new image " << char{t} << std::endl;
    dim3 dimBlock(32,32);
    dim3 dimGrid((Width  + dimBlock.x) / dimBlock.x,
                 (Height + dimBlock.y) / dimBlock.y);
    std::clog << "Render : " << t << std::endl;
    grayKernel<<<dimGrid, dimBlock>>>(Surface, Width, Height, t);
}

TEST(cuda_draw, basic_drawing) {
    window win(Width, Height, "Cuda Raytracer");
    auto w = win.getWindow();

    glfwSetKeyCallback(w, quit_with_q);
    glfwMakeContextCurrent(w);

    surface_raii vis(Width, Height);

    std::clog << "Init" << std::endl;
    float t = 0.f;
    while(!glfwWindowShouldClose(w)) {
        std::clog << "Loop" << std::endl;
        t += 0.5f;
        invokeRenderingKernel(vis.getSurface(), t);

        vis.render_gl_texture();

        glfwSwapBuffers(w);
        glfwPollEvents();
        std::clog << "Loop end" << std::endl;
    }
    input_manager::instance().clear();

    std::clog << "Done" << std::endl;
}

/// Write pixel data with cuda.
void render_cuda2(hipSurfaceObject_t& Surface, float t) {
    // Rendering
    invokeRenderingKernel(Surface, t);
}

TEST(cuda_draw, drawing_less_surfaces) {
    window win(Width, Height, "Cuda Raytracer");
    auto w = win.getWindow();

    glfwSetKeyCallback(w, quit_with_q);
    glfwMakeContextCurrent(w);

    surface_raii vis(Width, Height);

    float t = 0.f;
    while(!glfwWindowShouldClose(w)) {
        t += 0.5f;
        render_cuda2(vis.getSurface(), t);

        vis.render_gl_texture();

        glfwSwapBuffers(w);
        glfwWaitEvents();
    }
    input_manager::instance().clear();
    std::clog << "Done" << std::endl;
}

__global__ void black_kernel(hipSurfaceObject_t Surface, int Width, int Height) {
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    uchar4 BGColor;
    BGColor.x = 0;
    BGColor.y = 0;
    BGColor.z = 0;
    BGColor.w = 255;

    if(x < Width && y < Height)
        surf2Dwrite(BGColor, Surface, x * 4, y);
}

__global__ void trace_kernel(hipSurfaceObject_t Surface, const triangle* T, int Width, int Height) {
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    const float focal_length = 1.f;

    if(x < Width && y < Height)
    {
        ray R;
        R.origin    = coord{0.f, 0.f, -1.f};
        float DX = 2.f / ((float) Width  - 1);
        float DY = 2.f / ((float) Height - 1);
        R.direction = coord{x * DX - 1.f, y * DY - 1.f, focal_length};

        uchar4 FGColor;
        FGColor.x = 255;
        FGColor.y = 255;
        FGColor.z = 255;
        FGColor.w = 255;
        
        const auto Traced = R.intersects(*T);

        if(Traced.first) {
            surf2Dwrite(FGColor, Surface, x * 4, y);
        }
        //else {
            //surf2Dwrite(BGColor, Surface, x * 4, y);
        //}
    }
}

void raytrace_cuda(hipSurfaceObject_t& Surface, const triangle* T) {
    dim3 dimBlock(32,32);
    dim3 dimGrid((Width + dimBlock.x) / dimBlock.x,
                 (Height+ dimBlock.y) / dimBlock.y);
    trace_kernel<<<dimGrid, dimBlock>>>(Surface, T, Width, Height);
}

__global__ void trace_many_kernel(hipSurfaceObject_t Surface, 
                                  camera c,
                                  const triangle* Triangles, int TriangleCount,
                                  int Width, int Height)
{
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < Width && y < Height)
    {
        ray R = c.rayAt(x, y);

        uchar4 FGColor;
        FGColor.x = 255;
        FGColor.y = 255;
        FGColor.z = 255;
        FGColor.w = 255;

        uchar4 BGColor;
        BGColor.x = 0;
        BGColor.y = 0;
        BGColor.z = 0;
        BGColor.w = 255;

        triangle const* NearestTriangle = nullptr;
        intersect NearestIntersect;
        //NearestIntersect.depth = std::numeric_limits<float>::max;
        NearestIntersect.depth = 10000.f;

        // Find out the closes triangle
        for(std::size_t i = 0; i < TriangleCount; ++i)
        {
            const auto Traced = R.intersects(Triangles[i]);
            if(Traced.first)
            {
                if(Traced.second.depth < NearestIntersect.depth)
                {
                    NearestTriangle = &Triangles[i];
                    NearestIntersect = Traced.second;
                }
            }
        }

        if(NearestTriangle != nullptr) {
            FGColor.x = NearestIntersect.depth * 10.f;
            FGColor.y = NearestIntersect.depth * 10.f;
            FGColor.z = NearestIntersect.depth * 10.f;
            surf2Dwrite(FGColor, Surface, x * 4, y);
        }
        else {
            surf2Dwrite(BGColor, Surface, x * 4, y);
        }
    }

}

void raytrace_many_cuda(hipSurfaceObject_t& Surface, 
                        const camera& c,
                        const triangle* Triangles,
                        int TriangleCount) {
    dim3 dimBlock(32,32);
    dim3 dimGrid((c.width() + dimBlock.x) / dimBlock.x,
                 (c.height() + dimBlock.y) / dimBlock.y);
    trace_many_kernel<<<dimGrid, dimBlock>>>(Surface, c, Triangles, TriangleCount, 
                                             c.width(), c.height());
}

TEST(cuda_draw, drawing_traced_triangle) 
{
    window win(Width, Height, "Cuda Raytracer");
    auto w = win.getWindow();

    glfwSetKeyCallback(w, quit_with_q);
    glfwMakeContextCurrent(w);

    std::clog << "before surface creation" << std::endl;

    surface_raii vis(Width, Height);
    
    std::clog << "init" << std::endl;

    // Create the Triangle and Coordinates on the device
    thrust::device_vector<coord> Vertices(5);
    //Vertices[0] = {.5f,-1,1}; 
    //Vertices[1] = {-1,.5f,1};
    //Vertices[2] = {1,1,1};
    Vertices[0] = {0,-1,1}; 
    Vertices[1] = {-1,1,1};
    Vertices[2] = {1,1,1};
    Vertices[3] = {1,-0.8,1};
    Vertices[4] = {-1,0.8,1};

    const auto P0 = Vertices[0];
    const auto P1 = Vertices[1];
    const auto P2 = Vertices[2];
    const auto P3 = Vertices[3];
    const auto P4 = Vertices[4];

    thrust::device_vector<triangle> Triangles(3);
    Triangles[0] = {P0, P1, P2};
    Triangles[1] = {P0, P1, P3};
    Triangles[2] = {P4, P2, P0};
    std::clog << "triangles done" << std::endl;

    while(!glfwWindowShouldClose(w)) {
        dim3 dimBlock(32,32);
        dim3 dimGrid((Width + dimBlock.x) / dimBlock.x,
                     (Height+ dimBlock.y) / dimBlock.y);
        black_kernel<<<dimGrid, dimBlock>>>(vis.getSurface(), Width, Height);

        for(std::size_t i = 0; i < Triangles.size(); ++i)
        {
            const thrust::device_ptr<triangle> T = &Triangles[i];
            raytrace_cuda(vis.getSurface(), T.get());
        }

        vis.render_gl_texture();

        glfwSwapBuffers(w);
        glfwWaitEvents();
    } 
    input_manager::instance().clear();
    std::clog << "Done" << std::endl;
}

TEST(cuda_draw, draw_loaded_geometry)
{
    // Window stuff
    window win(Width, Height, "Cuda Raytracer");
    auto w = win.getWindow();

    glfwSetKeyCallback(w, quit_with_q);
    glfwSetCursorPosCallback(w, control_steering);
    glfwSetInputMode(w, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

    //c.lookAt({0.f, 0.f, 0.f});
    std::clog << c.steering() << std::endl;

    glfwMakeContextCurrent(w);

    // Cuda stuff
    surface_raii vis(Width, Height);

    // 3D Stuff
    world_geometry world("shapes.obj");
    std::clog << "initialized" << std::endl;

    const auto& Triangles = world.triangles();

    while(!glfwWindowShouldClose(w)) {
        dim3 dimBlock(32,32);
        dim3 dimGrid((Width + dimBlock.x) / dimBlock.x,
                     (Height + dimBlock.y) / dimBlock.y);
        black_kernel<<<dimGrid, dimBlock>>>(vis.getSurface(), Width, Height);

        raytrace_many_cuda(vis.getSurface(), c, 
                           Triangles.data().get(), Triangles.size());

        vis.render_gl_texture();

        glfwSwapBuffers(w);
        glfwWaitEvents();
    } 
    input_manager::instance().clear();
    std::clog << "Done" << std::endl;
}


int main(int argc, char** argv)
{
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
