#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "macros.h"
#include "triangle.h"
#include "ray.h"
#include "visualization.h"

#include <iostream>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_new.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <utility>


__global__ void grayKernel(hipSurfaceObject_t& Surface, int width, int height, float t)
{
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < width && y < height)
    {
        uchar4 Color;
        char new_t = t;
        Color.x = x - new_t;
        Color.y = y + new_t;
        Color.z = new_t;
        Color.w = 255;
        surf2Dwrite(Color, Surface, x * 4, y);
    }
}

void invokeRenderingKernel(hipSurfaceObject_t& Surface, float t)
{
    //std::cout << "Rendering new image " << char{t} << std::endl;
    dim3 dimBlock(32,32);
    dim3 dimGrid((640 + dimBlock.x) / dimBlock.x,
                 (480 + dimBlock.y) / dimBlock.y);
    grayKernel<<<dimGrid, dimBlock>>>(Surface, 640, 480, t);
}

/// Write pixel data with cuda.
void render_cuda(hipSurfaceObject_t& Surface, float t) {
    // Rendering
    invokeRenderingKernel(Surface, t);

    // raytracing should be something like that:
    // thrust::for_each(thrust::device, PrimaryRays.begin(), PrimaryRays.end(),
    // CUCALL [&CudaSurfaceObject,&Geometry](const ray& R) {
    //    // Determine all Intersections for that ray.
    //    thrust::device_vector<intersect> Hits;
    //    thrust::for_each(Geometry.begin(), Geometry.end(),
    //        [R,&Hits] (const triangle& T) {
    //            auto Test = R.intersects(T);
    //            if(Test.first) { Hits.push_back(Test.second); }
    //    });
    //    if(Hits.empty()) { 
    //        surf2Dwrite(BGColor, CudaSurfaceObject, R.u * 4, R.v);
    //    } 
    //    else {
    //        surf2Dwrite(FGColor, CudaSurfaceObject, R.u * 4, R.v);
    //    }
    // });


    //         // Determine the closest intersection of all Rays.
    //         auto Closest = *thrust::min_element(thrust::device, Hits.begin(), Hits.end(),
    //                         [](const intersect& I1, const intersect& I2) 
    //                         { return I1.deepth < I2.depth; });
    //         }
    //     });

    // Lulu
}

TEST(cuda_draw, basic_drawing) {
    visualization vis(640, 480);

    float t = 0.f;
    while(vis.looping()) {
        t += 0.1f;
        render_cuda(vis.getSurface(), t);
    }
}

/// Write pixel data with cuda.
void render_cuda2(hipSurfaceObject_t& Surface, float t) {
    // Rendering
    invokeRenderingKernel(Surface, t);
}

TEST(cuda_draw, drawing_less_surfaces) {
    visualization vis(640, 480);

    float t = 0.f;
    while(vis.looping()) {
        t += 0.1f;
        render_cuda2(vis.getSurface(), t);
    }
}


__global__ void trace_kernel(hipSurfaceObject_t Surface, triangle* T, int Width, int Height) {
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    const float focal_length = 2.f;

    if(x < Width && y < Height)
    {
        ray R;
        R.origin    = coord{0., 0., 0.};
        float DX = 2.f / ((float) Width  - 1);
        float DY = 2.f / ((float) Height - 1);
        R.direction = coord{x * DX - 1.f, y * DY - 1.f, focal_length};

        uchar4 FGColor;
        FGColor.x = 255;
        FGColor.y = 255;
        FGColor.z = 255;
        FGColor.w = 255;

        uchar4 BGColor;
        BGColor.x = 0;
        BGColor.y = 0;
        BGColor.z = 0;
        BGColor.w = 255;
        
        const auto Traced = R.intersects(*T);

        if(Traced.first) {
            surf2Dwrite(FGColor, Surface, x * 4, y);
        }
        else {
            surf2Dwrite(BGColor, Surface, x * 4, y);
        }
    }
}

void raytrace_cuda(hipSurfaceObject_t& Surface, triangle* T) {
    dim3 dimBlock(32,32);
    dim3 dimGrid((640 + dimBlock.x) / dimBlock.x,
                 (480 + dimBlock.y) / dimBlock.y);
    trace_kernel<<<dimGrid, dimBlock>>>(Surface, T, 640, 480);
}

TEST(cuda_draw, drawing_traced_triangle) 
{
    visualization vis(640, 480);

    // Create the Triangle and Coordinates on the device
    thrust::device_vector<coord> Vertices(3);
    //Vertices[0] = {.5f,-1,1}; 
    //Vertices[1] = {-1,.5f,1};
    //Vertices[2] = {1,1,1};
    Vertices[0] = {0,-1,1}; 
    Vertices[1] = {-1,1,1};
    Vertices[2] = {1,1,1};

    const thrust::device_ptr<coord> P0 = &Vertices[0];
    const thrust::device_ptr<coord> P1 = &Vertices[1];
    const thrust::device_ptr<coord> P2 = &Vertices[2];

    const auto triangle_void = thrust::device_malloc(sizeof(triangle));
    auto _ = gsl::finally([&triangle_void]() { thrust::device_free(triangle_void); });
    const auto triangle_ptr = thrust::device_new(triangle_void, 
                                                 triangle{P0.get(), P1.get(), P2.get()});

    while(vis.looping()) {
        raytrace_cuda(vis.getSurface(), triangle_ptr.get());
    }
}

int main(int argc, char** argv)
{
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
