#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "macros.h"
#include "triangle.h"
#include "ray.h"
#include "surface_raii.h"

#include <GLFW/glfw3.h>
#include <gsl/gsl>
#include <iostream>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_new.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <utility>


static void quit_with_q(GLFWwindow* w, int key, int scancode, int action, int mods)
{
    if(key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(w, GLFW_TRUE);
}


__global__ void grayKernel(hipSurfaceObject_t Surface, int width, int height, float t)
{
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < width && y < height)
    {
        uchar4 Color;
        char new_t = t;
        Color.x = x - new_t;
        Color.y = y + new_t;
        Color.z = new_t;
        Color.w = 255;
        surf2Dwrite(Color, Surface, x * 4, y);
    }
}

void invokeRenderingKernel(hipSurfaceObject_t& Surface, float t)
{
    //std::clog << "Rendering new image " << char{t} << std::endl;
    dim3 dimBlock(32,32);
    dim3 dimGrid((640 + dimBlock.x) / dimBlock.x,
                 (480 + dimBlock.y) / dimBlock.y);
    std::clog << "Render : " << t << std::endl;
    grayKernel<<<dimGrid, dimBlock>>>(Surface, 640, 480, t);
}

TEST(cuda_draw, basic_drawing) {
    auto Initialized = glfwInit();
    ASSERT_NE(Initialized, 0) << "Could not init glfw";

    gsl::owner<GLFWwindow*> w = glfwCreateWindow(640, 480, "Cuda Raytracer", nullptr, nullptr);
    glfwSetKeyCallback(w, quit_with_q);
    glfwMakeContextCurrent(w);

    surface_raii vis(640, 480);

    std::clog << "Init" << std::endl;
    float t = 0.f;
    while(!glfwWindowShouldClose(w)) {
        std::clog << "Loop" << std::endl;
        t += 0.1f;
        invokeRenderingKernel(vis.getSurface(), t);

        vis.render_gl_texture();

        glfwSwapBuffers(w);
        glfwPollEvents();
        std::clog << "Loop end" << std::endl;
    }

    std::clog << "Done" << std::endl;
    glfwDestroyWindow(w);
    glfwTerminate();
}

/// Write pixel data with cuda.
void render_cuda2(hipSurfaceObject_t& Surface, float t) {
    // Rendering
    invokeRenderingKernel(Surface, t);
}

TEST(cuda_draw, drawing_less_surfaces) {
    auto Initialized = glfwInit();
    ASSERT_NE(Initialized, 0) << "Could not init glfw";

    gsl::owner<GLFWwindow*> w = glfwCreateWindow(640, 480, "Cuda Raytracer", nullptr, nullptr);
    glfwSetKeyCallback(w, quit_with_q);
    glfwMakeContextCurrent(w);

    surface_raii vis(640, 480);

    float t = 0.f;
    while(!glfwWindowShouldClose(w)) {
        t += 0.1f;
        render_cuda2(vis.getSurface(), t);

        vis.render_gl_texture();

        glfwSwapBuffers(w);
        glfwWaitEvents();
    }
    std::clog << "Done" << std::endl;
    glfwDestroyWindow(w);
    glfwTerminate();
}

__global__ void black_kernel(hipSurfaceObject_t Surface, int Width, int Height) {
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    uchar4 BGColor;
    BGColor.x = 0;
    BGColor.y = 0;
    BGColor.z = 0;
    BGColor.w = 255;

    if(x < Width && y < Height)
        surf2Dwrite(BGColor, Surface, x * 4, y);
}

__global__ void trace_kernel(hipSurfaceObject_t Surface, triangle* T, int Width, int Height) {
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    const float focal_length = .5f;

    if(x < Width && y < Height)
    {
        ray R;
        R.origin    = coord{0., 0., 2.};
        float DX = 2.f / ((float) Width  - 1);
        float DY = 2.f / ((float) Height - 1);
        R.direction = coord{x * DX - 1.f, y * DY - 1.f, focal_length};

        uchar4 FGColor;
        FGColor.x = 255;
        FGColor.y = 255;
        FGColor.z = 255;
        FGColor.w = 255;

        
        const auto Traced = R.intersects(*T);

        if(Traced.first) {
            surf2Dwrite(FGColor, Surface, x * 4, y);
        }
        //else {
            //surf2Dwrite(BGColor, Surface, x * 4, y);
        //}
    }
}

void raytrace_cuda(hipSurfaceObject_t& Surface, triangle* T) {
    dim3 dimBlock(32,32);
    dim3 dimGrid((640 + dimBlock.x) / dimBlock.x,
                 (480 + dimBlock.y) / dimBlock.y);
    trace_kernel<<<dimGrid, dimBlock>>>(Surface, T, 640, 480);
}

TEST(cuda_draw, drawing_traced_triangle) 
{
    auto Initialized = glfwInit();
    ASSERT_NE(Initialized, 0) << "Could not init glfw";

    gsl::owner<GLFWwindow*> w = glfwCreateWindow(640, 480, "Cuda Raytracer", nullptr, nullptr);
    glfwSetKeyCallback(w, quit_with_q);
    glfwMakeContextCurrent(w);

    std::clog << "before surface creation" << std::endl;

    surface_raii vis(640, 480);
    
    std::clog << "init" << std::endl;

    // Create the Triangle and Coordinates on the device
    thrust::device_vector<coord> Vertices(5);
    //Vertices[0] = {.5f,-1,1}; 
    //Vertices[1] = {-1,.5f,1};
    //Vertices[2] = {1,1,1};
    Vertices[0] = {0,-1,1}; 
    Vertices[1] = {-1,1,1};
    Vertices[2] = {1,1,1};
    Vertices[3] = {1,-0.8,1};
    Vertices[4] = {-1,0.8,1};

    const thrust::device_ptr<coord> P0 = &Vertices[0];
    const thrust::device_ptr<coord> P1 = &Vertices[1];
    const thrust::device_ptr<coord> P2 = &Vertices[2];
    const thrust::device_ptr<coord> P3 = &Vertices[3];
    const thrust::device_ptr<coord> P4 = &Vertices[4];

    thrust::device_vector<triangle> Triangles(3);
    Triangles[0] = {P0.get(), P1.get(), P2.get()};
    Triangles[1] = {P0.get(), P1.get(), P3.get()};
    Triangles[2] = {P4.get(), P2.get(), P0.get()};
    std::clog << "triangles done" << std::endl;

    while(!glfwWindowShouldClose(w)) {
        dim3 dimBlock(32,32);
        dim3 dimGrid((640 + dimBlock.x) / dimBlock.x,
                     (480 + dimBlock.y) / dimBlock.y);
        black_kernel<<<dimGrid, dimBlock>>>(vis.getSurface(), 640, 480);

        std::clog << "loop" << std::endl;
        for(std::size_t i = 0; i < Triangles.size(); ++i)
        {
            std::clog << "trace" << std::endl;
            const thrust::device_ptr<triangle> T = &Triangles[i];
            raytrace_cuda(vis.getSurface(), T.get());
        }

        vis.render_gl_texture();

        glfwSwapBuffers(w);
        glfwWaitEvents();
    } 
    std::clog << "Done" << std::endl;
    glfwDestroyWindow(w);
    glfwTerminate();
}

int main(int argc, char** argv)
{
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
