#include "gtest/gtest.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/count.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_new.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/logical.h>
#include <thrust/transform.h>
#include <utility>

#include "ray.h"

constexpr std::size_t SquareDim = 10;

struct does_intersect {
    CUCALL bool operator()(const LIB::pair<bool, intersect>& r) { return r.first; };
};
struct has_good_depth {
    CUCALL bool operator()(const LIB::pair<bool, intersect>& r) 
    { return !r.first || (r.second.depth >= 1.f); };
};

struct fire_ray {
    CUCALL fire_ray(const thrust::device_ptr<triangle> T) : T{T} {}
    CUCALL ~fire_ray() = default;

    CUCALL LIB::pair<bool, intersect> operator()(const ray& Ray) 
    { return Ray.intersects(*T); }

    const thrust::device_ptr<triangle> T;
};

TEST(ray, init)
{
    ray R;
    R.origin = coord{0, 0, -15};
    R.direction = coord{0, 0, 1};
}

TEST(ray, intersection)
{
    ray R;
    R.origin    = coord{0, 0, 0};
    R.direction = coord{0, 0, 1};

    const coord P0{0, -10, 10}, P1{-10, 10, 10}, P2{10, 10, 10};
    triangle T{P0, P1, P2};

    bool DoesIntersect;
    intersect I;
    LIB::tie(DoesIntersect, I) = R.intersects(T);

    ASSERT_EQ(DoesIntersect, true) << "depth=" << I.depth;
    ASSERT_EQ(I.depth, 10.) << "(" << I.hit.x << "," << I.hit.y << "," << I.hit.z << ")\n" 
                            << "(" << I.normal.x << "," << I.normal.y << "," << I.normal.z << ")";
}

thrust::device_vector<ray> generateRays(const coord& Origin, std::size_t SquareDim) {
    // create multiple rays from the origin, 10x10 grid
    const float DY = 2.f / (SquareDim - 1);
    const float DX = 2.f / (SquareDim - 1);

    thrust::device_vector<ray> AllRays(SquareDim * SquareDim);
    std::size_t Index = 0;
    for(float Y = 1.f; Y > -1.f; Y-= DY)
    {
        for(float X = -1.f; X < 1.f; X+= DX)
        {
            const coord Dir{X, Y, 1.f};
            AllRays[Index] = ray{Origin, Dir};
            ++Index;
        }
    }
    return AllRays;
}

thrust::device_vector<LIB::pair<bool, intersect>> 
traceTriangle(const thrust::device_ptr<triangle> T, const thrust::device_vector<ray>& AllRays)
{
    OUT << "Before trace" << std::endl;
    // raytrace all the rays, and save result
    thrust::device_vector<LIB::pair<bool, intersect>> Result(AllRays.size());
    OUT << "Space for result allocated" << std::endl;

    LIB::transform(AllRays.begin(), AllRays.end(), 
                   Result.begin(), fire_ray{T});

    OUT << "Done tracing" << std::endl;
    return Result;
}

std::string bwOutput(const thrust::device_vector<LIB::pair<bool, intersect>>& Result, 
                     std::size_t SquareDim)
{
#if 0
    std::vector<std::pair<bool, intersect>> HostResult(Result.size());
    thrust::transform(Result.begin(), Result.end(), HostResult.begin(),
                      [] (const thrust::pair<bool, intersect>& R) {
                          return std::make_pair(R.first, R.second);
                      });
    OUT << "Data copied back" << std::endl;

    // output the data as "black white"
    std::size_t Index = 0;
    std::stringstream SS;
    for(std::size_t i = 0; i < SquareDim; ++i)
    {
        for(std::size_t j = 0; j < SquareDim; ++j)
        {
            bool DidHit;
            intersect I;
            std::tie(DidHit, I) = HostResult[Index];
            SS << (DidHit ? "*" : ".");
            ++Index;
        }
        SS << "\n";
    }

    OUT << "Done" << std::endl;
    return SS.str();
#else
    return "";
#endif
}

TEST(ray, trace_many_successfull)
{
    thrust::device_vector<coord> Vertices(4);
    Vertices[0] = {0,-1,1}; 
    Vertices[1] = {-1,1,1};
    Vertices[2] = {1,1,1};
    Vertices[3] = {0,0,2};
    const auto& P0 = Vertices[0];
    const auto& P1 = Vertices[1];
    const auto& P2 = Vertices[2];
    const auto& Origin = Vertices[3];

    triangle T{P0, P1, P2};
    const auto triangle_void = thrust::device_malloc(sizeof(triangle));
    const auto triangle_ptr = thrust::device_new(triangle_void, T);

    OUT << "Triangle and tracer origin created" << std::endl;

    const auto AllRays = generateRays(Origin, SquareDim);
    ASSERT_EQ(AllRays.size(), SquareDim * SquareDim);
    OUT << "Rays generated" << std::endl;
    const auto Result = traceTriangle(triangle_ptr, AllRays);
    ASSERT_EQ(Result.size(), SquareDim * SquareDim);
    OUT << "Raytracing done" << std::endl;

    const auto ContainsHit = LIB::any_of(thrust::device, Result.begin(), Result.end(), 
                                         does_intersect{});
    ASSERT_EQ(ContainsHit, true) << bwOutput(Result, SquareDim);

    const auto GoodDepth = LIB::all_of(thrust::device, Result.begin(), Result.end(),  
                                       has_good_depth{});
    ASSERT_EQ(GoodDepth, true) << bwOutput(Result, SquareDim);

    const auto HitCount = LIB::count_if(thrust::device, Result.begin(), Result.end(), 
                                        does_intersect{});
    ASSERT_GT(HitCount, 0.3 * SquareDim * SquareDim) << bwOutput(Result, SquareDim) +
                                                        "More hits are expected\n";
    ASSERT_LT(HitCount, 0.8 * SquareDim * SquareDim) << bwOutput(Result, SquareDim) +
                                                        "Less hits are expected\n";


    //std::cout << bwOutput(Result, SquareDim) << std::endl;
    OUT << "BW output done" << std::endl;
}

/*
TEST(ray, trace_many_failing)
{
    thrust::device_vector<coord> Vertices(4);
    Vertices[0] = {0,-1,1}; 
    Vertices[1] = {-1,1,1};
    Vertices[2] = {1,1,1};
    Vertices[3] = {0,0,2};
    const auto& P0 = Vertices[0];
    const auto& P1 = Vertices[1];
    const auto& P2 = Vertices[2];
    const auto& Origin = Vertices[3];

    triangle T{P0, P1, P2};

    const auto AllRays = generateRays(Origin, SquareDim);
    const auto Result = traceTriangle(T, AllRays);
    
    //std::cout << bwOutput(Result, SquareDim) << std::endl;
    const auto ContainsHit = LIB::any_of(Result.begin(), Result.end(), does_intersect{});
    ASSERT_EQ(ContainsHit, false) << bwOutput(Result, SquareDim);
}
*/

int main(int argc, char** argv) {
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS(); 
}
