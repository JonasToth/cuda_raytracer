#include "surface_raii.h"

#include <chrono>
#include <gsl/gsl>
#include <iostream>

#pragma diagnostic push
#pragma GCC diagnostic ignored "-Wignored-qualifiers"
#include <png-plusplus/image.hpp>
#include <png-plusplus/rgba_pixel.hpp>
#include <stdexcept>
#include <thread>

surface_raii::surface_raii(int width, int height, render_target target)
  : __target{target}
  , __width{width}
  , __height{height}
  , __texture{0}
{
    __initialize_render_target();
    __initialize_cuda_surface();
}

surface_raii::~surface_raii()
{
    if (__target == render_target::texture) {
        // Destroy the opengl texture
        glDeleteTextures(1, &__texture);
        // Destroy link between cuda and opengl
        hipGraphicsUnmapResources(1, &__cuda_resource);
    }

    hipDestroySurfaceObject(__cuda_surface);
}

namespace
{
png::image<png::rgba_pixel> memory_to_png(const std::vector<uint8_t>& memory,
                                          std::size_t width, std::size_t height,
                                          const int channels)
{
    png::image<png::rgba_pixel> img(width, height);
    for (std::size_t y = 0ul; y < height; ++y) {
        for (std::size_t x = 0ul; x < width; ++x) {
            const auto idx = channels * (y * width + x);
            const png::rgba_pixel pixel(memory[idx], memory[idx + 1], memory[idx + 2]);
            // Otherwise its upside down, because opengl
            img.set_pixel(x, height - y - 1, pixel);
        }
    }
    return img;
}
} // namespace

void surface_raii::save_as_png(const std::string& file_name) const
{
    const auto memory = __get_texture_memory();
    // const not allowed, IDK why
    auto img = memory_to_png(memory, __width, __height, __channels);
    img.write(file_name);
}

void surface_raii::__initialize_render_target()
{
    // https://stackoverflow.com/questions/19244191/
    // cuda-opengl-interop-draw-to-opengl-texture-with-cuda
    if (__target == render_target::texture)
        __initialize_opengl_texture();
    else if (__target == render_target::memory)
        __initialize_memory_texture();
    else
        throw std::logic_error{"Unexcpected value for render_target!"};
}


void surface_raii::__initialize_opengl_texture()
{
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &__texture);

    if (__texture == 0)
        throw std::runtime_error{"Could not create opengl texture"};

    glBindTexture(GL_TEXTURE_2D, __texture);
    { // beauty stuff for opengl, maybe skip?
        Expects(__channels == 4);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, __width, __height, 0, GL_RGBA,
                     GL_UNSIGNED_BYTE, nullptr);
    }
    glBindTexture(GL_TEXTURE_2D, 0);

    const auto E = hipGraphicsGLRegisterImage(&__cuda_resource, __texture, GL_TEXTURE_2D,
                                               hipGraphicsRegisterFlagsWriteDiscard);

    // error checking on the cuda call
    switch (E) {
        case hipErrorInvalidDevice:
            throw std::runtime_error{"Cuda bind texture: invalid device"};
        case hipErrorInvalidValue:
            throw std::runtime_error{"Cuda bind texture: invalid value"};
        case hipErrorInvalidHandle:
            throw std::runtime_error{"Cuda bind texture: invalid resource handle"};
        case hipErrorUnknown:
            throw std::runtime_error{"Cuda bind texture: unknown error"};
        default:
            break;
    }

    // Memory mapping
    hipGraphicsMapResources(1, &__cuda_resource);
}

void surface_raii::__initialize_memory_texture()
{
    // Allocate memory in RAM
    __memory_texture.resize(__channels * __width * __height);
}

void surface_raii::__initialize_cuda_surface()
{
    // source: Internet :)
    // https://stackoverflow.com/questions/19244191/
    // cuda-opengl-interop-draw-to-opengl-texture-with-cuda
    // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#surface-object-api
    hipGraphicsSubResourceGetMappedArray(&__cuda_array, __cuda_resource, 0, 0);
    __cuda_array_resource_desc.resType = hipResourceTypeArray;
    __cuda_array_resource_desc.res.array.array = __cuda_array;

    // Surface creation
    hipCreateSurfaceObject(&__cuda_surface, &__cuda_array_resource_desc);
}

void surface_raii::render_gl_texture() noexcept
{
    glBindTexture(GL_TEXTURE_2D, __texture);
    {
        glBegin(GL_QUADS);
        {
            glTexCoord2f(0.0f, 0.0f);
            glVertex2f(-1.0f, -1.0f);
            glTexCoord2f(1.0f, 0.0f);
            glVertex2f(+1.0f, -1.0f);
            glTexCoord2f(1.0f, 1.0f);
            glVertex2f(+1.0f, +1.0f);
            glTexCoord2f(0.0f, 1.0f);
            glVertex2f(-1.0f, +1.0f);
        }
        glEnd();
    }
    glBindTexture(GL_TEXTURE_2D, 0);
    glFinish();
}

std::vector<uint8_t> surface_raii::__get_texture_memory() const
{
    if (__target == render_target::texture) {
        std::vector<uint8_t> gl_texture_data(__width * __height * __channels);
        glReadPixels(0, 0, __width, __height, GL_RGBA, GL_UNSIGNED_BYTE,
                     gl_texture_data.data());

        return gl_texture_data;
    } else if (__target == render_target::memory)
        return __memory_texture;
    else
        throw std::logic_error{"Unexpected render target"};
}

#pragma GCC diagnostic pop
