#include "surface_raii.h"
#include <chrono>
#include <iostream>
#include <stdexcept>
#include <thread>


surface_raii::surface_raii(int width, int height)
    : __width{width}
    , __height{height}
    , __texture{0}
    , __cuda_array{}
    , __cuda_array_resource_desc{}
    , __cuda_resource{}
    , __cuda_surface{}
{
    __initialize_texture();
    __initialize_cuda_surface();
}

surface_raii::~surface_raii() 
{
    std::clog << "Destroying the surface and texture" << std::endl;
    // Destroy the opengl texture
    glDeleteTextures(1, &__texture);

    // Destroy all cuda and opengl connections
    hipDestroySurfaceObject(__cuda_surface);
    hipGraphicsUnmapResources(1, &__cuda_resource);

}

// https://stackoverflow.com/questions/19244191/cuda-opengl-interop-draw-to-opengl-texture-with-cuda
void surface_raii::__initialize_texture() {
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &__texture);

    if(__texture == 0)
        throw std::runtime_error{"Could not create opengl texture"};

    glBindTexture(GL_TEXTURE_2D, __texture);
    { // beauty stuff for opengl, maybe skip?
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, __width, __height, 0, GL_RGBA, 
					 GL_UNSIGNED_BYTE, nullptr);
    }
    glBindTexture(GL_TEXTURE_2D, 0);

    const auto E = hipGraphicsGLRegisterImage(&__cuda_resource, __texture, GL_TEXTURE_2D, 
                                               hipGraphicsRegisterFlagsWriteDiscard);

    // error checking on the cuda call
    switch (E) {
        case hipErrorInvalidDevice: throw std::runtime_error{"Cuda bind texture: invalid device"};
        case hipErrorInvalidValue: throw std::runtime_error{"Cuda bind texture: invalid value"};
        case hipErrorInvalidHandle: throw std::runtime_error{"Cuda bind texture: invalid resource handle"};
        case hipErrorUnknown: throw std::runtime_error{"Cuda bind texture: unknown error"};
        default: break;
    }

    // Memory mapping
    hipGraphicsMapResources(1, &__cuda_resource); 
}

void surface_raii::__initialize_cuda_surface()
{
    // source: Internet :)
    // https://stackoverflow.com/questions/19244191/cuda-opengl-interop-draw-to-opengl-texture-with-cuda
    hipGraphicsSubResourceGetMappedArray(&__cuda_array, __cuda_resource, 0, 0);

    __cuda_array_resource_desc.resType = hipResourceTypeArray;
    __cuda_array_resource_desc.res.array.array = __cuda_array;

    // Surface creation
    hipCreateSurfaceObject(&__cuda_surface, &__cuda_array_resource_desc); 
}

void surface_raii::render_gl_texture() noexcept
{
    glBindTexture(GL_TEXTURE_2D, __texture);
    {
        glBegin(GL_QUADS);
        {
            glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
            glTexCoord2f(1.0f, 0.0f); glVertex2f(+1.0f, -1.0f);
            glTexCoord2f(1.0f, 1.0f); glVertex2f(+1.0f, +1.0f);
            glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, +1.0f); 
        }
        glEnd();
    }
    glBindTexture(GL_TEXTURE_2D, 0);
    glFinish();
}
